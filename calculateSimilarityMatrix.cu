
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_WIDTH 16

//calculate similarity matrix using global memory
__global__ void calculateSimilarityMatrixGlobal(float *M,float *P,int width,int height,float *avgArray,float *norm_val) 
{
	//thread indices and block indices
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;

        //row and column indices of element of P being calculated
        int row = by * blockDim.y + ty;
        int column = bx * blockDim.x  + tx;

	float val = 0.0;
	float num,denom; 
	for(int i=0;i<width;i++)
	{
		if(row<height && column<height && M[row*width+i]>0.0f && M[column*width+i]>0.0f)
			val += (M[row*width+i]-avgArray[row])*(M[column*width+i]-avgArray[column]);
			
	}
	if(row<height && column<height && norm_val[row]>0.0f && norm_val[column]>0.0f)
		denom = (float)sqrt(norm_val[row])*sqrt(norm_val[column]);
	if(row<height && column< height)
	{	
		if(denom>0.0f)
			P[row*height+column] = val/denom;
		else
			P[row*height+column] = 0.0f;
	}
}


// Matrix multiplication kernel thread specification
__global__ void calculateSimilarityMatrixNoTranspose(float *M,float *N,float *P,int width, int height,float *avgArray, float* norm_val)
{

  //variables declared in shared memory
        __shared__ float Ms[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Ns[TILE_WIDTH][TILE_WIDTH];
        //thread indices and block indices
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;

        //row and column indices of element of P being calculated
        int row = by * TILE_WIDTH + ty;
        int column = bx * TILE_WIDTH + tx;

        float p_Val = 0;
        float numer, denom;

        // compute target element value
        for(int i=0;i<ceilf(width/(float)TILE_WIDTH);i++){

                if(row < height && (i*TILE_WIDTH + tx)<width)
                        Ms[ty][tx] = M[row*width + i*TILE_WIDTH + tx];
                else
                        Ms[ty][tx] = 0.0;

                //if(i*TILE_WIDTH+threadIdx.y <height && column<width)
                //        Ns[threadIdx.y][threadIdx.x] = N[(i*TILE_WIDTH+threadIdx.y)*width+column];
                //else
                //        Ns[threadIdx.y][threadIdx.x] = 0.0;


                //ensure that all values of the tile is available
                __syncthreads();

                for(int j=0;j<TILE_WIDTH;j++){
                        if(row < height && column < height && Ms[ty][j] > 0.0f && Ns[j][tx] > 0.0f){
                                p_Val += (Ms[ty][j]-avgArray[row]) * (Ns[j][tx]-avgArray[column]);
                                //p_Val += (Ms[ty][j]) * (Ms[tx][j]);
                        }
                }
                __syncthreads();

                if(column < height && row < height && norm_val[row] > 0.0f && norm_val[column] > 0.0f){
                        denom = (float) sqrt(norm_val[row]) * sqrt(norm_val[column]);
                }
                //ensure that all values of the tile are used
                __syncthreads();
        }

        //check the boundary condition
        if(row < height && column < height)
        {
                if(denom>0.0f)
                        P[row*height+column] = p_Val/denom;
                else
                        P[row*height+column] = 0.0f;
        }

}





// Matrix multiplication kernel thread specification
__global__ void calculateSimilarityMatrix(float *M,float *N,float *P,int width, int height,float *avgArray, float* norm_val)
{

  //variables declared in shared memory
        __shared__ float Ms[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Ns[TILE_WIDTH][TILE_WIDTH];
        //thread indices and block indices
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int bx = blockIdx.x;
        int by = blockIdx.y;

        //row and column indices of element of P being calculated
        int row = by * TILE_WIDTH + ty;
        int column = bx * TILE_WIDTH + tx;

        float p_Val = 0;
        float numer, denom;

        // compute target element value
        for(int i=0;i<ceilf(width/(float)TILE_WIDTH);i++){
               
                if(row < height && (i*TILE_WIDTH + tx)<width)
                        Ms[ty][tx] = M[row*width + i*TILE_WIDTH + tx];
                else
                        Ms[ty][tx] = 0.0;
		
		if(i*TILE_WIDTH+threadIdx.y <width && column<height)
                        Ns[threadIdx.y][threadIdx.x] = N[(i*TILE_WIDTH+threadIdx.y)*height+column];
                else
                        Ns[threadIdx.y][threadIdx.x] = 0.0;


                //ensure that all values of the tile is available
                __syncthreads();

                for(int j=0;j<TILE_WIDTH;j++){
                        if(row < height && column < height && Ms[ty][j] > 0.0f && Ns[j][tx] > 0.0f){
                                p_Val += (Ms[ty][j]-avgArray[row]) * (Ns[j][tx]-avgArray[column]);
                                //p_Val += (Ms[ty][j]) * (Ms[tx][j]);
                        }
                }
		__syncthreads();

                if(column < height && row < height && norm_val[row] > 0.0f && norm_val[column] > 0.0f){
                        denom = (float) sqrt(norm_val[row]) * sqrt(norm_val[column]);
                }
                //ensure that all values of the tile are used
                __syncthreads();
        }

        //check the boundary condition
        if(row < height && column < height)
	{
		if(denom>0.0f)
               		P[row*height+column] = p_Val/denom;
		else
			P[row*height+column] = 0.0f;
	}

}
