#include "hip/hip_runtime.h"
#include<stdio.h>
#include "getAverageAndNorm.cu"
#include "dataReader.c"
#include "calculateSimilarityMatrix.cu"
#include "transpose_kernel.cu"

float avg[ROWS];
float norm_val[ROWS];
float sim[ROWS*ROWS];
float global_sim[ROWS*ROWS];
// float transpose[COLS][ROWS];

int isSimilarityCorrect(){
	printf("%d\t%d\n",ROWS,COLS);
        for(int i = 0; i < ROWS; i++){
                for(int j = i; j < ROWS; j++){
			float temp = similarity_matrix[i][j];
                        if(abs(sim[i*ROWS+j] - temp) > 0.01){
                                printf("(%d, %d): GPU=%f CPU=%f\n", i, j, sim[i*ROWS+j], similarity_matrix[i*ROWS+j]);
                                return 0;
                        }
                }
        }
        return 1;
}

int isAverageCorrect(){
        for(int i = 0; i < ROWS; i++){
                if(fabs(avg[i] - avg_rating[i]) > 0.00001){
                        printf("(%d): GPU=%f CPU=%f\n", i, avg[i], avg_rating[i]);
                        return 0;
                }
        }
        return 1;
}

// void transposeMatrix(){
//         for(int i = 0; i < ROWS; i++){
//                 for(int j = 0; j < COLS; j++){
//                         transpose[j][i] = ratings[i][j];
//                 }
//         }
// }

int main(){
        float *d_ratings, *d_avg, *d_norm, *d_sim, *d_transpose, *d_sim_global;
        dim3 dimGrid(ROWS/TILE_WIDTH + 1, ROWS/TILE_WIDTH + 1, 1);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
	printf("Loading CSV data.........\n");
        hipEventRecord(start);
        readCSV();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Time taken to load csv = %f seconds\n\n",(float)milliseconds/1000);
	printf("Executing serial code\n");
	printf("Computing average serial code\n");
	hipEventRecord(start);
	serial_mean();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
        milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Time taken for computing average serial implementation = %f seconds\n\n",(float)milliseconds/1000);

	printf("computing serial similarity matrix\n\n");
	hipEventRecord(start);
	build_similarity_matrix();
	hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Time taken for Similarity matrix serial implementation = %f seconds\n\n",(float)milliseconds/1000);
        printf("Allocating device memory and copy data\n");
        hipEventRecord(start);
        hipMalloc((void**)&d_ratings, ROWS * COLS * sizeof(float));
        hipMalloc((void**)&d_avg, ROWS * sizeof(float));
        hipMalloc((void**)&d_norm, ROWS * sizeof(float));
        hipMalloc((void**)&d_sim, ROWS * ROWS * sizeof(float));
	hipMalloc((void**)&d_transpose, ROWS * COLS * sizeof(float));
        hipMemcpy(d_ratings, ratings, ROWS * COLS * sizeof(float), hipMemcpyHostToDevice);


        //transpose kernel being called
        // Matrix out = AllocateMatrix(4, 6, 1);
//     Matrix d_out = AllocateDeviceMatrix(out);
//     CopyToDeviceMatrix(d_out, out);


        hipMemcpy(sim, d_sim, ROWS * ROWS * sizeof(float), hipMemcpyDeviceToHost);
	//transpose<<<transGrid, transBlock>>>(d_transpose, d_ratings, COLS, ROWS);
        float trans_blocks_x = COLS/TILE_DIM+1;
        float trans_blocks_y = ROWS/TILE_DIM+1;

        dim3 transGrid(trans_blocks_x, trans_blocks_y);
        dim3 transBlock(TILE_DIM, TILE_DIM);

	//transpose<<<transGrid, transBlock>>>(d_transpose, d_ratings, COLS, ROWS);
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventSynchronize(stop);
        milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Time taken for allocating device memory and loading data = %f seconds\n\n",(float)milliseconds/1000);

        hipMalloc((void**)&d_sim_global, ROWS * ROWS * sizeof(float));
	
	printf("Computing transpose.....\n");
	hipEventRecord(start);
	transpose<<<transGrid, transBlock>>>(d_transpose, d_ratings, COLS, ROWS);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds,start,stop);
	printf("Time taken to compute transpose = %f seconds\n\n",(float)milliseconds/1000);	

        printf("Getting Average and Norm.....\n");
        hipEventRecord(start);
	GetAverageAndNorm<<<1, ROWS>>>(d_transpose, ROWS, COLS, d_avg, d_norm);
        hipDeviceSynchronize();
	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);
        hipMemcpy(avg, d_avg, ROWS * sizeof(float), hipMemcpyDeviceToHost);
	milliseconds = 0.0f;
	hipEventElapsedTime(&milliseconds,start,stop);
	printf("Time taken for computing average and norm = %f seconds\n\n",(float)milliseconds/1000);

	printf("Computing Similarity matrix using tiling\n");
	hipEventRecord(start);
        calculateSimilarityMatrix<<<dimGrid, dimBlock>>>(d_ratings,d_transpose,d_sim,COLS, ROWS, d_avg, d_norm);
        hipMemcpy(sim, d_sim, ROWS * ROWS * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	milliseconds = 0.0f;
	hipDeviceSynchronize();
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Execution time for Tiled parallel version similarity matrix computation: %f  seconds\n\n",(float)milliseconds/1000);

	printf("Computing Similarity Matrix using Global Memory\n");
        hipEventRecord(start);
	calculateSimilarityMatrixGlobal<<<dimGrid, dimBlock>>>(d_ratings, d_sim_global, COLS, ROWS, d_avg, d_norm);
        hipMemcpy(global_sim, d_sim_global, ROWS * ROWS * sizeof(float), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        milliseconds = 0.0f;
	hipDeviceSynchronize();
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Execution time for global memory version similarity matrix computation : %f  seconds\n\n",(float)milliseconds/1000);
        printf("Average values are %s\n", isAverageCorrect()? "correct" : "incorrect");
        printf("Similarity values are %s\n", isSimilarityCorrect()? "correct" : "incorrect");

}
