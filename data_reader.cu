
#include <hip/hip_runtime.h>
#include<stdio.h>
//#include "getAvgAndNorm.cu"

//TODO: Modify for larger dataset. For small dataset, actual no. of rows were greater than what was specified in the MovieLens website

#define ROWS 629
#define COLS 9000

char fileName[] = "./ratings_small.csv";
float ratings[ROWS][COLS];
float avg[ROWS];
float average[ROWS];
float norm_val[ROWS];
void readCSV(){
        int user, item;
        float score;
        long ts;
        FILE *fp;
        fp = fopen(fileName, "r");
        fscanf(fp, "%*[^\n]\n", NULL);
        do
				{
                fscanf(fp,"%d,%d,%f,%ld\n", &user, &item, &score, &ts);
                ratings[user][item] = score;
        }
	while(!feof(fp));
}

void serial_mean(){
        int row,col;
        for(row=0;row<ROWS;row++){
                average[row] = 0.0;
                int count = 0;
                for(col=0;col<COLS;col++){
                        if(ratings[row][col]!=0.0)
                        {
                                count++;
                                average[row] += ratings[row][col];
                        }
                }
                average[row] = (float)average[row]/count;
        }
}

//serial implementation of computing Ri-R_mean
int compare()
{
        int row,col;
        for(row=0;row<ROWS;row++){
               if(fabs(average[row] - avg[row]) > 0.5)
                        return 0;
        }
        return 1;
}
__global__ void GetAverageAndNorm(float *R, int N, float *avg, float *norm){
        int tid = blockIdx.x*blockDim.x + threadIdx.x, countNonZero = 0.0f;
        float sum = 0, avgThread;
        for(int i = 0; i < N; i++){
                if (R[tid * N + i] > 0.0f) {
                        sum += R[tid * N + i];
                        countNonZero++;
                }
        }
        avgThread = (float) sum/countNonZero;
        avg[tid] = avgThread;
	sum = 0;
        for(int i = 0;  i < N; i++){
                if (R[tid * N + i] != 0) {
                        float t = R[tid * N + i] - avgThread;
                        sum += t*t;
                }
        }
	norm[tid] = sum;
}

int main(){
       	float *d_ratings, *d_avg, *d_norm;
        readCSV();
                             
        hipMalloc((void**)&d_ratings, ROWS * COLS * sizeof(float));
        hipMalloc((void**)&d_avg, ROWS * sizeof(float));
        hipMalloc((void**)&d_norm, ROWS * sizeof(float));
        hipMemcpy(d_ratings, ratings, ROWS * COLS * sizeof(float), hipMemcpyHostToDevice);
        GetAverageAndNorm<<<1, ROWS>>>(d_ratings, COLS, d_avg, d_norm);
        hipMemcpy(avg, d_avg, ROWS * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(norm_val, d_norm, ROWS * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < ROWS; i++){
		printf("%f ", avg[i]);
        } 
        for(int i = 0; i < ROWS; i++){
		printf("%f ", norm_val[i]);
        } 
        serial_mean();  
        printf("\n\n\n\n\n RESULT = %d", compare()); 


}
