
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void GetAverageAndNorm(float *R, int cols, int rows, float *avg, float *norm){
   /* int tid = blockIdx.x*blockDim.x + threadIdx.x, countNonZero = 0;
    float sum = 0.0f, avgThread = 0.0f;
    for(int i = 0; i < cols; i++){
            if (R[tid * cols + i] > 0.0f) {
                    sum += R[tid * cols + i];
                    countNonZero++;
            }
    }
    if(countNonZero > 0)
        avgThread = (float) sum/countNonZero;
    else
        avgThread = 0.0f;
    if(tid < rows)
        avg[tid] = avgThread;
    sum = 0;
    for(int i = 0;  i < cols; i++){
        if (R[tid * cols + i] > 0.0f){
            float t = R[tid * cols + i] - avgThread;
            sum += t*t;
        }
    }
    if(tid < rows)
        norm[tid] = sum;*/
int tid = blockIdx.x*blockDim.x + threadIdx.x, countNonZero = 0;
    float sum = 0.0f, avgThread = 0.0f;
    for(int i = 0; i < rows; i++){
            if (R[i * cols + tid] > 0.0f) {
                    sum += R[i * cols + tid];
                    countNonZero++;
            }
    }
    if(countNonZero > 0)
        avgThread = (float) sum/countNonZero;
    else
        avgThread = 0.0f;
    if(tid < cols)
        avg[tid] = avgThread;
    sum = 0;
    for(int i = 0;  i < rows; i++){
        if (R[i * cols + tid] > 0.0f){
            float t = R[i * cols + tid] - avgThread;
            sum += t*t;
        }
    }
    if(tid < cols)
        norm[tid] = sum;
}