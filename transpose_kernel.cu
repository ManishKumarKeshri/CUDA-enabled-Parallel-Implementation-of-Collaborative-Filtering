
#include <hip/hip_runtime.h>
#include<stdio.h>

#define TILE_DIM 32

__global__ void transpose(float *odata, const float *idata,
int width, int height)
{
__shared__ float tile[TILE_DIM][TILE_DIM+1];
int x = blockIdx.x * TILE_DIM + threadIdx.x;
int y = blockIdx.y * TILE_DIM + threadIdx.y;

if (x < width && y < height)
{ tile[threadIdx.y][threadIdx.x] = idata[y*width + x]; }
__syncthreads();

x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
y = blockIdx.x * TILE_DIM + threadIdx.y;
if (y < width && x < height)
{ odata[y*height + x] = tile[threadIdx.x][threadIdx.y]; }
}
