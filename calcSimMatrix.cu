#include "hip/hip_runtime.h"
#ifndef _SIM_MATRIX_CALCULATION_
#define _SIM_MATRIX_CALCULATION_

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define TILE_WIDTH 16

__global__ naiveSimMatrix(float **R,float **S, float * avgArray, int width,int numUsers){
	int tid = threadIdx.x;
	int row = blockDim.y*blockIdx.y+threadIdx.y;
	int col = blockDim.x*blockIdx.x+threadIdx.y;
	float val = 0.0;
	for(int i=0;i<width;i++) {
		val += (R[row*width+i]-avgArray[row])*(R[col*width+i]-avgArray[col]);
		sigmaA += R[row*width+i]*R[row*width+i];
		sigmaB += R[col*width+i]*R[col*width+i];
	}
	float denom = sqrt(sigmaA*sigmaB)
	S[row*width+col] = val;
}

__global__ void calcSimMatrix(float *R,float *S,float *aveArray,int width)
{
  int bx = blockIdx.x; //get x and y coordinates of block
	int by = blockIdx.y;
	int tx = threadIdx.x; //get x and y coordinates of thread
	int ty = threadIdx.y;

  if(by>bx) // skip the calculation of results below diagonal
    return;

  __shared__ float ATile[TILE_WIDTH][TILE_WIDTH];
	__shared__ float BTile[TILE_WIDTH][TILE_WIDTH];
  __shared__ float AAverage[TILE_WIDTH];
	__shared__ float BAverage[TILE_WIDTH];

  if(ty==0){
    // Prasanth's function to load average values of ATile into AAverage
  }
  if(ty==1){
    //same here to load average values of BTile into BAverage
  }
  int row = by * TILE_WIDTH + ty; // get row and col in S
  int col = bx * TILE_WIDTH + tx;

  float dotproduct = 0, sigmA = 0, sigmB = 0, pearson_correlation;
  if(row<TILE_WIDTH && col<TILE_WIDTH){ // check the bounds
    for(int i=0;i<ceilf(width/(float)TILE_WIDTH);i++){ // loop all the elements in the row
      ATile[tx][ty] = R[row * width + i * TILE_WIDTH + tx] - AAverage[ty];
      BTile[tx][ty] = R[col + (i * TILE_WIDTH + ty) * width] - BAverage[ty];

      __syncthreads();

      for(int k=0;k<TILE_WIDTH; k++){
        dotproduct += ATile[ty][k] * BTile[tx][k];
        sigmA += ATile[ty][k] * ATile[ty][k];
        sigmB += BTile[tx][k] * BTile[tx][k];
      }

      __syncthreads();
    }
    pearson_correlation = dotproduct / sqrt(sigmA * sigmB);
    S[row * width + col] = pearson_correlation;
  }
}

#endif
